#include "hip/hip_runtime.h"
// -*- c++ -*-

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR
#include <stdio.h>
#include <algorithm>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include "util.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "mortoncoder.h"
#include <bitset>

using namespace hipcub;
//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------
bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory
//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/*
struct Pair
{
    float   key;
    int     value;
    bool operator<(const Pair &b) const
    {
        if (key < b.key)
            return true;
        if (key > b.key)
            return false;
        // Return true if key is negative zero and b.key is positive zero
        unsigned int key_bits   = SafeBitCast<unsigned int>(key);
        unsigned int b_key_bits = SafeBitCast<unsigned int>(b.key);
        unsigned int HIGH_BIT   = 1u << 31;
        return ((key_bits & HIGH_BIT) != 0) && ((b_key_bits & HIGH_BIT) == 0);
    }
};



void Initialize(
    float           *h_keys,
    int             *h_values,
    float           *h_reference_keys,
    int             *h_reference_values,
    int             num_items)
{
    Pair *h_pairs = new Pair[num_items];
    for (int i = 0; i < num_items; ++i)
    {
        RandomBits(h_keys[i]);
        RandomBits(h_values[i]);
        h_pairs[i].key    = h_keys[i];
        h_pairs[i].value  = h_values[i];
    }
    if (g_verbose)
    {
        printf("Input keys:\n");
        DisplayResults(h_keys, num_items);
        printf("\n\n");
        printf("Input values:\n");
        DisplayResults(h_values, num_items);
        printf("\n\n");
    }
    std::stable_sort(h_pairs, h_pairs + num_items);
    for (int i = 0; i < num_items; ++i)
    {
        h_reference_keys[i]     = h_pairs[i].key;
        h_reference_values[i]   = h_pairs[i].value;
    }
    delete[] h_pairs;
}

*/
/*

https://stackoverflow.com/a/17401122
__device__ 
static float atomicMax(float* address, float val)
{
  int* address_as_i = (int*) address;
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = ::atomicCAS(address_as_i, assumed,
		      __float_as_int(::fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ 
static float atomicMin(float* address, float val)
{
  int* address_as_i = (int*) address;
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = ::atomicCAS(address_as_i, assumed,
		      __float_as_int(::fminf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}
*/

// Refer following for morton encoding.
//https://www.forceflow.be/2013/10/07/morton-encodingdecoding-through-bit-interleaving-implementations/
// https://github.com/Forceflow/libmorton  - This is a good library for CPUs
__device__ 
void splitBy3(int a, uint64_t& s) {
    s = a & 0x1fffff;
    s = (s | s << 32) & 0x1f00000000ffff;
    s = (s | s << 16) & 0x1f0000ff0000ff;
    s = (s | s << 8) & 0x100f00f00f00f00f;
    s = (s | s << 4) & 0x10c30c30c30c30c3;
    s = (s | s << 2) & 0x1249249249249249;
}




__global__
void compute64BitMortonCode3dPoint(float* centroids, int count, int* bboxint, uint64_t* mortonCodes)
{
  //printf("here1\n");
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int nt = blockDim.x*gridDim.x;
  
  for (int i = tid; i < count; i = i + nt) {

    uint64_t s_x;
    uint64_t s_y;
    uint64_t s_z;

    int cx_int = __float2int_rn(max((centroids[3*i] - bboxint[0])*4194303/(bboxint[3] - bboxint[0]), 0.0));
    cx_int = cx_int <= 4194303 ? cx_int : 4194303;   // This is just to make sure there is no numerical error 
    int cy_int = __float2int_rn(max((centroids[3*i+1] - bboxint[1])*4194303/(bboxint[4] - bboxint[1]), 0.0));
    cy_int = cy_int <= 4194303 ? cy_int : 4194303;
    int cz_int = __float2int_rn(max((centroids[3*i+2] - bboxint[2])*4194303/(bboxint[5] - bboxint[2]), 0.0));
    cz_int = cz_int <= 4194303 ? cz_int : 4194303;
    
    splitBy3(cx_int, s_x);
    splitBy3(cy_int, s_y);
    splitBy3(cz_int, s_z);
    mortonCodes[i] = s_x | s_y << 1 | s_z << 2 ;
    
  }
  
  
}



__global__
void computeTriangleCentroids(float* d_vertices, int* d_indices, float* d_centroids, int nFaces, int* bbox) 
{
  
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int nt = blockDim.x*gridDim.x;
  
  for(int i = tid; i < nFaces; i = i+nt) { 
    int idx1 = d_indices[3*i + 0];
    int idx2 = d_indices[3*i + 1];
    int idx3 = d_indices[3*i + 2];

    float x1 = d_vertices[3*idx1 + 0];
    float y1 = d_vertices[3*idx1 + 1];
    float z1 = d_vertices[3*idx1 + 2];

    float x2 = d_vertices[3*idx2 + 0];
    float y2 = d_vertices[3*idx2 + 1];
    float z2 = d_vertices[3*idx2 + 2];

    float x3 = d_vertices[3*idx3 + 0];
    float y3 = d_vertices[3*idx3 + 1];
    float z3 = d_vertices[3*idx3 + 2];
    
    float c_x = (x1 + x2 + x3)/3.0;
    float c_y = (y1 + y2 + y3)/3.0;
    float c_z = (z1 + z2 + z3)/3.0;

    d_centroids[3*i+0] = c_x;
    d_centroids[3*i+1] = c_y;
    d_centroids[3*i+2] = c_z;

    
    atomicMin(&bbox[0], __float2int_rd(c_x));
    atomicMin(&bbox[1], __float2int_rd(c_y)); 
    atomicMin(&bbox[2], __float2int_rd(c_z)); 

    atomicMax(&bbox[3], __float2int_ru(c_x));
    atomicMax(&bbox[4], __float2int_ru(c_y)); 
    atomicMax(&bbox[5], __float2int_ru(c_z));
    
  }
 
}

#include <cassert>
#include <cmath>
//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------
__host__
void copyTinyObjSceneToGPU(tinyobj::attrib_t& attrib, std::vector<tinyobj::shape_t>& shapes, float* &d_vertices, int* &d_indices, int &nFaces, hipError_t &error) {
			   //std::vector<tinyobj::material_t>& meterials = NULL) {
  // For now maybe just copy vertices and face indices for only first shape ?
  // They will fire me for writing this code.

  error = hipMalloc(&d_vertices, attrib.vertices.size()*sizeof(float));
  if(error) return;
  error = hipMemcpy(d_vertices, attrib.vertices.data(), attrib.vertices.size()*sizeof(float), hipMemcpyHostToDevice);
  if(error) return;

  int n = shapes[0].mesh.indices.size();
  assert (n%3 == 0);
  nFaces = n/3;
  std::vector<int> vertex_indices(n);
  
  std::transform(shapes[0].mesh.indices.cbegin(), shapes[0].mesh.indices.cend(), vertex_indices.cbegin(),
		 vertex_indices.begin(), [](tinyobj::index_t idx, int i) -> int { return idx.vertex_index; });

  error = hipMalloc(&d_indices, n*sizeof(int));
  if(error) return;
  error = hipMemcpy(d_indices, vertex_indices.data(), n*sizeof(int), hipMemcpyHostToDevice);
  if(error) return;
  
  return;
}




hipError_t cuda_kernel(tinyobj::attrib_t attrib, std::vector<tinyobj::shape_t> shapes)
{

  float* d_vertices;
  int* d_indices;
  int nFaces = -1;
  hipError_t error = hipSuccess;

  
  error = hipMalloc(&d_vertices, attrib.vertices.size()*sizeof(float));
  if(error) return error;
  error = hipMemcpy(d_vertices, attrib.vertices.data(), attrib.vertices.size()*sizeof(float), hipMemcpyHostToDevice);
  if(error) return error;

  copyTinyObjSceneToGPU(attrib, shapes, d_vertices, d_indices, nFaces, error);
  if(error) return error;

  int blockSize;      // The launch configurator returned block size 
  int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
  int gridSize;       // The actual grid size needed, based on input size 

  error = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeTriangleCentroids, 0, nFaces); 
  if(error) return error;

  std::cout << "Suggested block Size: " << blockSize << "\n";
  std::cout << "Suggested mingrid Size: " << minGridSize << "\n";

  float* d_centroids;
  error = hipMalloc(&d_centroids, 3*nFaces*sizeof(float));
  if(error) {
    std::cout << hipGetErrorString(error);
    std::cout <<  "exit from here\n";
    return error;
  }

  int* bbox;
  error = hipMalloc(&bbox, 6*sizeof(int));
  if(error) {
    std::cout <<  "exit from here22\n";
    return error;
  }
  int inftyup = std::numeric_limits<int>::max();
  int inftydown = std::numeric_limits<int>::min();
  int* bboxstart = new int[6];
  bboxstart[0] = inftyup;
  bboxstart[1] = inftyup;
  bboxstart[2] = inftyup;
  bboxstart[3] = inftydown;
  bboxstart[4] = inftydown;
  bboxstart[5] = inftydown;

  error = hipMemcpy(bbox, bboxstart, 6*sizeof(int), hipMemcpyHostToDevice);

  if(error) {
    std::cout <<  "exit from here2\n";
    return error;
  }
  dim3 dimGrid(minGridSize,1,1);
  dim3 dimBlock(blockSize,1,1);

  hipDeviceSynchronize();

  error = hipGetLastError();
  if(error) {
    std::cout << hipGetErrorString(error) << " before first\n";
    return error;
  }
  std::cout << "centroid nfaces " << nFaces << "\n";
  computeTriangleCentroids<<<32, 64>>>(d_vertices, d_indices, d_centroids, nFaces, bbox);
  hipDeviceSynchronize();

  error = hipGetLastError();
  if(error) {
    std::cout << hipGetErrorString(error) << " first\n";
    return error;
  }

  int* hostbbox = (int*)malloc(6*sizeof(int));
  hipMemcpy(hostbbox, bbox, 6*sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "bbox : " << hostbbox[0] << ","<< hostbbox[1] << ","<< hostbbox[2] << ","<< hostbbox[3] << ","<< hostbbox[4] << ","<< hostbbox[5] << "\n";

  uint64_t* mortonCodes;
  error = hipMalloc(&mortonCodes, nFaces*sizeof(uint64_t));
  if(error) return error;

  error = hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, compute64BitMortonCode3dPoint, 0, nFaces); 
  if(error) return error;
  dim3 dimGrid2(32,1,1);
  dim3 dimBlock2(32,1,1);

  hipDeviceSynchronize();
  compute64BitMortonCode3dPoint<<<dimGrid2, dimBlock2>>>(d_centroids, nFaces, bbox, mortonCodes);
  hipDeviceSynchronize();

  error = hipGetLastError();
  if(error) {
    std::cout << hipGetErrorString(error) << " over here\n";
    return error;
  }
  
  
  float* centroids = (float*)malloc(3*nFaces*sizeof(float));
  hipMemcpy(centroids, d_centroids, 3*nFaces*sizeof(float), hipMemcpyDeviceToHost);
  uint64_t* mortonhost = (uint64_t*)malloc(nFaces*sizeof(uint64_t));
  hipMemcpy(mortonhost, mortonCodes, nFaces*sizeof(uint64_t), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  for(int i = 0; i < 100; i++) {
    int idx1 = shapes[0].mesh.indices[i].vertex_index;
    int idx2 = shapes[0].mesh.indices[i+1].vertex_index;
    int idx3 = shapes[0].mesh.indices[i+2].vertex_index;
    
    float x1 = attrib.vertices[3*idx1 +0];
    float y1 = attrib.vertices[3*idx1 +1];
    float z1 = attrib.vertices[3*idx1 +2];

    float x2 = attrib.vertices[3*idx2 +0];
    float y2 = attrib.vertices[3*idx2 +1];
    float z2 = attrib.vertices[3*idx2 +2];

    float x3 = attrib.vertices[3*idx3 +0];
    float y3 = attrib.vertices[3*idx3 +1];
    float z3 = attrib.vertices[3*idx3 +2];


    std::cout << x1 << " " << x2 << " " << x3 << y1 << " " << y2 << " " << y3 << z1 << " " << z2 << " " << z3 << "\n";
    std::cout << centroids[3*i]  << " " << centroids[3*i + 1]  << " " << centroids[3*i+2] << " " << std::bitset<64>(mortonhost[i]) << "\n";
  }
  




  /*
  float time;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  int blockSize;      // The launch configurator returned block size 
  int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
  int gridSize;       // The actual grid size needed, based on input size 

  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeTriangleCentroids, 0, nFaces); 

  std::cout << "minGridSize: " << minGridSize << " blockSize: " << blockSize << "\n";
  */

  //computeTriangleCentroids(d_v, d_indices, centroid, )
  
  



  
  /*
    int num_items = 150;
    // Initialize command line
    printf("hipcub::DeviceRadixSort::SortPairs() %d items (%d-byte keys %d-byte values)\n",
        num_items, int(sizeof(float)), int(sizeof(int)));
    fflush(stdout);
    // Allocate host arrays
    float   *h_keys             = new float[num_items];
    float   *h_reference_keys   = new float[num_items];
    int     *h_values           = new int[num_items];
    int     *h_reference_values = new int[num_items];
    // Initialize problem and solution on host

    
    Initialize(h_keys, h_values, h_reference_keys, h_reference_values, num_items);


    
    // Allocate device arrays
    DoubleBuffer<float> d_keys;
    DoubleBuffer<int>   d_values;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(float) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(float) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(int) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(int) * num_items));
    // Allocate temporary storage
    size_t  temp_storage_bytes  = 0;
    void    *d_temp_storage     = NULL;
    CubDebugExit(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    // Initialize device arrays
    CubDebugExit(hipMemcpy(d_keys.d_buffers[d_keys.selector], h_keys, sizeof(float) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_values.d_buffers[d_values.selector], h_values, sizeof(int) * num_items, hipMemcpyHostToDevice));
    // Run
    CubDebugExit(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items));
    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference_keys, d_keys.Current(), num_items, true, g_verbose);
    printf("\t Compare keys (selector %d): %s\n", d_keys.selector, compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    compare = CompareDeviceResults(h_reference_values, d_values.Current(), num_items, true, g_verbose);
    printf("\t Compare values (selector %d): %s\n", d_values.selector, compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    // Cleanup
    if (h_keys) delete[] h_keys;
    if (h_reference_keys) delete[] h_reference_keys;
    if (h_values) delete[] h_values;
    if (h_reference_values) delete[] h_reference_values;
    if (d_keys.d_buffers[0]) CubDebugExit(g_allocator.DeviceFree(d_keys.d_buffers[0]));
    if (d_keys.d_buffers[1]) CubDebugExit(g_allocator.DeviceFree(d_keys.d_buffers[1]));
    if (d_values.d_buffers[0]) CubDebugExit(g_allocator.DeviceFree(d_values.d_buffers[0]));
    if (d_values.d_buffers[1]) CubDebugExit(g_allocator.DeviceFree(d_values.d_buffers[1]));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    printf("\n\n");

    */
    return error;
}
